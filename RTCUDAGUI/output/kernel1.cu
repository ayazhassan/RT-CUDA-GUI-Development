#include "hip/hip_runtime.h"
__global__ void matrix_add(float *C,float const *__restrict__ A,float const *__restrict__ B,int N){
	int tid=threadIdx.x;
	int bid=blockIdx.x;
	int ij=bid*matrix_addBLOCKSIZE+tid;
	{
		int i=(ij/N)*matrix_addMERGE_LEVEL;
		int j=(ij%N)*matrix_addSKEW_LEVEL;
		for(int m=0;m<matrix_addMERGE_LEVEL;m++)
			for(int n=0;n<matrix_addSKEW_LEVEL;n++)
				C[((i+m))*N+((j+n))]=A[((i+m))*N+((j+n))]+B[((i+m))*N+((j+n))];
				
	}
}
