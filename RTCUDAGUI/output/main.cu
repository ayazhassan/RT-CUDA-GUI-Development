#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
void checkCudaError(const char *msg)
{
        hipError_t err = hipGetLastError();
        if(hipSuccess != err){
                printf("%s(%i) : CUDA error : %s : (%d) %s\n", __FILE__, __LINE__, msg, (int)err, hipGetErrorString(err));
                exit (-1);
        }
}
#include "rcuda.h"
#include "kernel1params.h"
#include "kernel1.cu"
#include "kernel2params.h"
#include "kernel2.cu"

int main(int argc,char *argv[]){
	int N=1024;
	int GPU=2;
	if(argc>1)N=atoi(argv[1]);
	
	if(argc>2)GPU=atoi(argv[2]);
	
	hipSetDevice (GPU);
	float *A,*B,*C;
	int memsize=N*N*sizeof(float );
	hipMallocManaged(&A,memsize);
	
	hipMallocManaged(&B,memsize);
	
	hipMallocManaged(&C,memsize);
	
	dim3 threads(matrix_addBLOCKSIZE,1);
	dim3 grid(N*N/matrix_addBLOCKSIZE/matrix_addMERGE_LEVEL/matrix_addSKEW_LEVEL,1);
	matrix_add<<<grid,threads>>>(C,A,B,N);
	hipDeviceSynchronize();
	
	checkCudaError("matrix_add error: ");
	
	dim3 threads(matrix_subBLOCKSIZE,1);
	dim3 grid(N*N/matrix_subBLOCKSIZE/matrix_subMERGE_LEVEL/matrix_subSKEW_LEVEL,1);
	matrix_sub<<<grid,threads>>>(C,A,B,N);
	hipDeviceSynchronize();
	
	checkCudaError("matrix_sub error: ");
	
	hipFree (A);
	hipFree (B);
	hipFree (C);
	hipDeviceReset();
	
}
